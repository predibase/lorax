#include "hip/hip_runtime.h"
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>

#include "sgmv_cutlass.cuh"

template bool sgmv<nv_half>(nv_half *y, nv_half *x, nv_half **w,
                            int32_t *s_start, int32_t *s_end,
                            void *tmp_d, int num_problems, int d_in, int d_out,
                            int layer_idx);

template bool sgmv<hip_bfloat16>(hip_bfloat16 *y, hip_bfloat16 *x, hip_bfloat16 **w,
                                int32_t *s_start, int32_t *s_end,
                                void *tmp_d, int num_problems, int d_in, int d_out,
                                int layer_idx);
