#include "hip/hip_runtime.h"
#include "rope.cuh"
#include "util.cuh"
#include "matrix_view.cuh"

const int THREADS_X = 32;
const int THREADS_Y = 4;
const int MAX_POS_EMBEDDINGS = 32768;   // Actual number doesn't matter
const int MAX_ROWS = 32768;             // Actual number doesn't matter

typedef void (*fp_rope_cuda_kernel)
(
    half*,
    const half*,
    const half*,
    int,
    int,
    int,
    int,
    const int32_t*,
    int
);

template<bool use_half2>
__global__ void rope_cuda_kernel
(
    half* __restrict__ x,
    const half* __restrict__ sin,
    const half* __restrict__ cos,
    int rows_per_batch,
    int head_dim,
    int num_heads,
    int past_len,
    const int32_t* __restrict__ past_lens,
    int threads_y
)
{
    MatrixView_half_rw x_(x, MAX_ROWS, head_dim);
    MatrixView_half sin_(sin, MAX_POS_EMBEDDINGS, head_dim);
    MatrixView_half cos_(cos, MAX_POS_EMBEDDINGS, head_dim);

    int column = (blockIdx.x * THREADS_X + threadIdx.x); if constexpr (use_half2) column *= 2;
    int half_dim = head_dim / 2;
    if (column >= half_dim) return;

    int row = blockIdx.y * threads_y + threadIdx.y;
    if (row >= rows_per_batch) return;
    int batch_offset = blockIdx.z * rows_per_batch;
    int row_offset = batch_offset + row;

    // Get sin and cos

    if (past_len == -1)
    {
        past_len = past_lens[blockIdx.z];
        past_len = max(past_len, 0);
    }
    else if (past_lens)
    {
        past_len += past_lens[blockIdx.z];
    }

    int sincos_row = past_len + row / num_heads;
    sincos_row = max(sincos_row, 0);

    if constexpr (use_half2)
    {
        half2 cos2_l = cos_.item_half2(sincos_row, column);
        half2 cos2_r = cos_.item_half2(sincos_row, column + half_dim);
        half2 sin2_l = sin_.item_half2(sincos_row, column);
        half2 sin2_r = sin_.item_half2(sincos_row, column + half_dim);
        sin2_l = __hneg2(sin2_l);

        // Apply embedding to row

        half2 item2_l = x_.item_half2(row_offset, column);
        half2 item2_r = x_.item_half2(row_offset, column + half_dim);
        half2 item2_ls = __hmul2(item2_r, sin2_l);
        half2 item2_rs = __hmul2(item2_l, sin2_r);
        item2_l = __hfma2(item2_l, cos2_l, item2_ls);
        item2_r = __hfma2(item2_r, cos2_r, item2_rs);
        x_.set_half2(row_offset, column, item2_l);
        x_.set_half2(row_offset, column + half_dim, item2_r);
    }
    else
    {
        half cos_l = cos_.item(sincos_row, column);
        half cos_r = cos_.item(sincos_row, column + half_dim);
        half sin_l = sin_.item(sincos_row, column);
        half sin_r = sin_.item(sincos_row, column + half_dim);
        sin_l = __hneg(sin_l);

        // Apply embedding to row

        half item_l = x_.item(row_offset, column);
        half item_r = x_.item(row_offset, column + half_dim);
        half item_ls = __hmul(item_r, sin_l);
        half item_rs = __hmul(item_l, sin_r);
        item_l = __hfma(item_l, cos_l, item_ls);
        item_r = __hfma(item_r, cos_r, item_rs);
        x_.set(row_offset, column, item_l);
        x_.set(row_offset, column + half_dim, item_r);
    }
}

fp_rope_cuda_kernel pick_rope_cuda_kernel(bool use_half2)
{
    if (use_half2) return rope_cuda_kernel<true>;
    else           return rope_cuda_kernel<false>;
};

void rope_cuda
(
    half* x,
    const half* sin,
    const half* cos,
    const int batch_size,
    const int rows_per_batch,
    const int head_dim,
    const int num_heads,
    const int past_len,
    const int32_t* past_lens
)
{
    bool use_half2 = true;

    // For large batch sizes we risk exceeding grid dimension of 65535, so shift to block dimension instead

    int threads_y = THREADS_Y;
    while (DIVIDE(rows_per_batch, threads_y) > 65535) threads_y *= 2;

    dim3 blockDim, gridDim;
    blockDim.x = THREADS_X;
    blockDim.y = threads_y;
    gridDim.x = DIVIDE(head_dim / (use_half2 ? 2 : 1), THREADS_X);
    gridDim.y = DIVIDE(rows_per_batch, threads_y);
    gridDim.z = batch_size;

    fp_rope_cuda_kernel kernel = pick_rope_cuda_kernel(use_half2);
    kernel<<<gridDim, blockDim>>>(x, sin, cos, rows_per_batch, head_dim, num_heads, past_len, past_lens, threads_y);

    cuda_check( hipPeekAtLastError() );
}
