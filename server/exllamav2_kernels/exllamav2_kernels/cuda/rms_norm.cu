#include "hip/hip_runtime.h"
#include "rms_norm.cuh"
#include "util.cuh"

#if defined(USE_ROCM)
#define __shfl_xor_sync(mask, var, laneMask) __shfl_xor(var, laneMask)
#define NUM_WARPS (1024 / warpSize)
#define WARP_SIZE (warpSize)
#else
#define NUM_WARPS 32
#define WARP_SIZE 32
#endif

// y = x * w / sqrt(row_mean(x * x) + epsilon)

#define BLOCK_SIZE WARP_SIZE
#define NUM_THREADS (NUM_WARPS * WARP_SIZE)

typedef void (*fp_rms_norm_kernel)
(
    const half*,
    const half*,
    half*,
    const float,
    const float,
    const int,
    const int
);

template <int blocks_per_warp>
__global__ void rms_norm_kernel
(
    const half* __restrict__ x,
    const half* __restrict__ w,
    half* __restrict__ y,
    const float epsilon,
    const float r_dim,
    const int rows,
    const int dim
)
{
    int warp_id = threadIdx.x / WARP_SIZE;
    int lane_id = threadIdx.x % WARP_SIZE;
    int row = blockIdx.x;
    const half* x_row = x + row * dim;
    half* y_row = y + row * dim;

    //int blocks_per_warp = DIVIDE(dim, NUM_THREADS);

    // Compute sum of squares for each block

    float sum = 0.0f;
    float itemf[blocks_per_warp];

    #pragma unroll
    for (int i = 0; i < blocks_per_warp; i++)
    {
        int column = warp_id * WARP_SIZE + lane_id + NUM_THREADS * i;
        if (column >= dim) break;

        float f = __half2float(x_row[column]);
        f = fmaxf(-65504.0f, fminf(f, 65504.0f));
        itemf[i] = f;
        sum = fma(f, f, sum);
    }

    // Shuffle to sum across lanes

    __shared__ float sums[NUM_WARPS];

    for(int offset = warpSize / 2; offset > 0; offset /= 2) sum += __shfl_xor_sync(0xffffffff, sum, offset);
    if (lane_id == 0) sums[warp_id] = sum;
    __syncthreads();

    // Load partial sums from across warps, shuffle again across lanes

    sum = sums[lane_id];
    for(int offset = warpSize / 2; offset > 0; offset /= 2) sum += __shfl_xor_sync(0xffffffff, sum, offset);

    // Get norm

    float rmf = rsqrtf(sum * r_dim + epsilon);

    // Normalize x, scaling by w

    #pragma unroll 4
    for (int i = 0; i < blocks_per_warp; i++)
    {
        int column = warp_id * WARP_SIZE + lane_id + NUM_THREADS * i;
        if (column >= dim) return;

        float x_itemf = itemf[i];
        float w_itemf = __half2float(w[column]);
        float n = x_itemf * w_itemf * rmf;
        y_row[column] = __float2half_rn(n);
    }
}

fp_rms_norm_kernel pick_rms_norm_kernel(const int blocks_per_warp)
{
    if (blocks_per_warp == 1) return rms_norm_kernel<1>;
    if (blocks_per_warp == 2) return rms_norm_kernel<2>;
    if (blocks_per_warp == 3) return rms_norm_kernel<3>;
    if (blocks_per_warp == 4) return rms_norm_kernel<4>;
    if (blocks_per_warp == 5) return rms_norm_kernel<5>;
    if (blocks_per_warp == 6) return rms_norm_kernel<6>;
    if (blocks_per_warp == 7) return rms_norm_kernel<7>;
    if (blocks_per_warp == 8) return rms_norm_kernel<8>;
	return NULL;
}


void rms_norm_cuda
(
    const half* x,
    const half* w,
    half* y,
    const float epsilon,
    const int rows,
    const int dim
)
{
    dim3 blockDim, gridDim;
    blockDim.x = NUM_THREADS;
    blockDim.y = 1;
    gridDim.x = rows;
    gridDim.y = 1;

    float r_dim = 1.0f / (float) dim;

    int blocks_per_warp = DIVIDE(dim, NUM_THREADS);
    fp_rms_norm_kernel kernel = pick_rms_norm_kernel(blocks_per_warp);
    kernel<<<gridDim, blockDim>>>(x, w, y, epsilon, r_dim, rows, dim);
}
